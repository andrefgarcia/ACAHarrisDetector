#include "hip/hip_runtime.h"

// Based on CUDA SDK template from NVIDIA

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <unistd.h>

// includes, project
#include <cutil_inline.h>

#define max(a,b) (((a)>(b))?(a):(b))
#define min(a,b) (((a)<(b))?(a):(b))



// harris detector code to run on the host
void harrisDetectorHost(unsigned int *h_idata, unsigned int w, unsigned int h, 
                int ws,               // window size
                int threshold,        // threshold value to detect corners
                unsigned int * reference)
{
    int i,j,k,l;  // indexes in image
    int Ix, Iy;   // gradient in XX and YY
    int R;        // R metric
    int sumIx2, sumIy2, sumIxIy;

    for(i=0; i<h; i++) //height image
    {
        for(j=0; j<w; j++) //width image
        {
            reference[i*w+j]=h_idata[i*w+j]/4; // to obtain a faded background image
        }
    }

    for(i=ws+1; i<h-ws; i++) //height image
    {
        for(j=ws+1; j<w-ws; j++) //width image
        {
           sumIx2=0.0;sumIy2=0.0;sumIxIy=0.0;
           for(k=-ws; k<=ws; k++) //height window
              {
                  for(l=-ws; l<=ws; l++) //width window
                  {
                        Ix = ((int)h_idata[(i+k-1)*w + j+l] - (int)h_idata[(i+k)*w + j+l])/32;         
                        Iy = ((int)h_idata[(i+k)*w + j+l-1] - (int)h_idata[(i+k)*w + j+l])/32;         
			sumIx2 += Ix*Ix;
			sumIy2 += Iy*Iy;
			sumIxIy += Ix*Iy;
                  }
              }

              R = sumIx2*sumIy2-sumIxIy*sumIxIy-0.05*(sumIx2+sumIy2)*(sumIx2+sumIy2);
              if(R > threshold) {
                   reference[i*w+j]=255; 
              }
        }
    }
}   

// harris detector code to run on the GPU
void harrisDetectorDevice(unsigned int *h_idata, unsigned int w, unsigned int h, 
                  unsigned int ws, unsigned int threshold, 
                  unsigned int * h_odata)
{
    //TODO
}

// print command line format
void usage(char *command) 
{
    printf("Usage: %s [-h] [-d device] [-i inputfile] [-o outputfile] [-r referenceFile] [-w windowsize] [-t threshold]\n",command);
}

// main
int main( int argc, char** argv) 
{

    // default command line options
    int deviceId = 0;
    char *fileIn=(char *)"chess.pgm",*fileOut=(char *)"chessOut.pgm",*referenceOut=(char *)"reference.pgm";
    int ws = 2, threshold = 500;

    // parse command line arguments
    int opt;
    while( (opt = getopt(argc,argv,"d:i:o:r:w:t:h")) !=-1)
    {
        switch(opt)
        {

            case 'd':
                if(sscanf(optarg,"%d",&deviceId)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;

            case 'i':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }

                fileIn = strdup(optarg);
                break;
            case 'o':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                fileOut = strdup(optarg);
                break;
            case 'r':
                if(strlen(optarg)==0)
                {
                    usage(argv[0]);
                    exit(1);
                }
                referenceOut = strdup(optarg);
                break;
            case 'w':
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&ws)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 't':
                if(strlen(optarg)==0 || sscanf(optarg,"%d",&threshold)!=1)
                {
                    usage(argv[0]);
                    exit(1);
                }
                break;
            case 'h':
                usage(argv[0]);
                exit(0);
                break;

        }
    }

    // select cuda device
    cutilSafeCall( hipSetDevice( deviceId ) );
    
    // create events to measure host harris detector time and device harris detector time

    hipEvent_t startH, stopH, startD, stopD;
    hipEventCreate(&startH);
    hipEventCreate(&stopH);
    hipEventCreate(&startD);
    hipEventCreate(&stopD);



    // allocate host memory
    unsigned int* h_idata=NULL;
    unsigned int h,w;

    //load pgm
    if (cutLoadPGMi(fileIn, &h_idata, &w, &h) != CUTTrue) {
        printf("Failed to load image file: %s\n", fileIn);
        exit(1);
    }

    // allocate mem for the result on host side
    unsigned int* h_odata = (unsigned int*) malloc( h*w*sizeof(unsigned int));
    unsigned int* reference = (unsigned int*) malloc( h*w*sizeof(unsigned int));
 
    // detect corners at host

    hipEventRecord( startH, 0 );
    harrisDetectorHost(h_idata, w, h, ws, threshold, reference);   
    hipEventRecord( stopH, 0 ); 
    hipEventSynchronize( stopH );

    // detect corners at GPU
    hipEventRecord( startD, 0 );
    harrisDetectorDevice(h_idata, w, h, ws, threshold, h_odata);   
    hipEventRecord( stopD, 0 ); 
    hipEventSynchronize( stopD );
    
    // check if kernel execution generated and error
    cutilCheckMsg("Kernel execution failed");

    float timeH, timeD;
    hipEventElapsedTime( &timeH, startH, stopH );
    printf( "Host processing time: %f (ms)\n", timeH);
    hipEventElapsedTime( &timeD, startD, stopD );
    printf( "Device processing time: %f (ms)\n", timeD);

    // save output images
    if (cutSavePGMi(referenceOut, reference, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", referenceOut);
        exit(1);
    }
    if (cutSavePGMi(fileOut, h_odata, w, h) != CUTTrue) {
        printf("Failed to save image file: %s\n", fileOut);
        exit(1);
    }

    // cleanup memory
    cutFree( h_idata);
    free( h_odata);
    free( reference);

    cutilDeviceReset();
}
